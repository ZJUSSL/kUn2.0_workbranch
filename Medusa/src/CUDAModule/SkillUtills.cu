#include <hip/hip_runtime.h>



#include <math.h>
#include <stdio.h>

#define FRAME_PERIOD (1 / 20.0)
#define ZERO_NUM (1e-8)
#define A_FACTOR (1.5)
#define PI (3.14159265359)
#define G (9.8)
#define SQRT_2 (1.414)
#define PLAYER_SIZE (9.0)

#define PLAYER_CENTER_TO_BALL_CENTER (60)
#define MAX_PLAYER (16)
#define THREAD_NUM_PASS (128)
#define BLOCK_X_PASS (16)
#define BLOCK_Y_PASS (MAX_PLAYER * 2)
#define MAX_BALL_SPEED (6500)
#define MIN_BALL_SPEED (1000)
#define BALL_SPEED_UNIT ((MAX_BALL_SPEED - MIN_BALL_SPEED) / BLOCK_X_PASS)

//#define MAX_CHIP_SPEED (400)
//#define MIN_CHIP_SPEED (50)
//#define CHIP_SPEED_UNIT ((MAX_CHIP_SPEED - MIN_CHIP_SPEED) / BLOCK_X_PASS)

#define MIN_DELTA_TIME (0)
#define OUR_RESPONSE_TIME (0.0)
#define THEIR_RESPONSE_TIME (0.0)
#define CAN_NOT_GET_STOP_BALL (true)

//场地参数
#define PITCH_LENGTH (12000)
#define PITCH_WIDTH (9000)
#define PENALTY_LENGTH (1200)
#define PENALTY_WIDTH (2400)

//时间预测的运动学参数
#define OUR_MAX_SPEED (3000)
#define OUR_MAX_ACC (4500)
#define OUR_MAX_DEC (4500)

#define THEIR_MAX_SPEED (3000)
#define THEIR_MAX_ACC (4500)
#define THEIR_MAX_DEC (4500)

//#define CHIP_ENERGY_LEFT_1 (0.34) //挑球后与地面碰撞后的能量剩余比例
//#define CHIP_ENERGY_LEFT_2 (0.97)

//计算点位评分
//#define BLOCK_X_FOR_POS_SCORE (4)
//#define BLOCK_Y_FOR_POS_SCORE (4)
//#define THREAD_X_FOR_POS_SCORE (32)
//#define THREAD_Y_FOR_POS_SCORE (32)
#define INITIAL_VALUE (99999)

// 评估函数各项的阈值
//__constant__ float maxDistToGoal =  900000;//sqrt(pow(PITCH_LENGTH, 2) + pow(PITCH_WIDTH, 2))
//__constant__ float minShootAngle = 0;
//__constant__ float maxShootAngle = PI / 2;
//__constant__ float maxDistToBall = 900000;//sqrt(pow(PITCH_LENGTH, 2) + pow(PITCH_WIDTH, 2))
//__constant__ float minDistToPassLine = 10.0;
//__constant__ float maxDistToPassLine = 900000;//sqrt(pow(PITCH_LENGTH, 2) + pow(PITCH_WIDTH, 2))
//__constant__ float minDistToEnemy = 30.0;
//__constant__ float maxDistToEnemy = 900000;//sqrt(pow(PITCH_LENGTH, 2) + pow(PITCH_WIDTH, 2))
// 评估函数各项的权重
//__constant__ float weight1 = 5;// 1.距离对方球门的距离
//__constant__ float weight2 = 8;// 2.射门有效角度
//__constant__ float weight3 = 0.5;// 3.跟球的距离
//__constant__ float weight4 = 0.3;// 4.对方车到传球线的距离
//__constant__ float weight5 = 0.5;// 5.对方车到接球点的距离
// 挑球模型参数
#define CHIP_FIRST_ANGLE (54.29 / 180.0 * PI)
#define CHIP_SECOND_ANGLE (45.59 / 180.0 * PI)
#define CHIP_LENGTH_RATIO (1.266)
#define CHIP_VEL_RATIO  (0.6372)
#define MIN_CHIP_DIST (0.5)
#define MAX_CHIP_DIST (4.0)
#define MAX_CHIP_SPEED (50.0 * sqrt(2*G*MAX_CHIP_DIST/sin(2*CHIP_FIRST_ANGLE)))
#define MIN_CHIP_SPEED (50.0 * sqrt(2*G*MIN_CHIP_DIST/sin(2*CHIP_FIRST_ANGLE)))
#define CHIP_SPEED_UNIT ((MAX_CHIP_SPEED - MIN_CHIP_SPEED) / BLOCK_X_PASS)
typedef struct {
    float x, y;
} Vector;

typedef struct {
    float x, y;
} Point;

typedef struct {
    Point Pos;
    Vector Vel;
    bool isValid;
} Player;

typedef struct {
    Point interPos;
    float interTime;
    float Vel;
    float dir;
    int playerIndex;
    float deltaTime;
    float Q;
} rType;

typedef struct {
    Point p;
    float score;
} scoreAndPoint;

__device__ float Min(float a, float b) { return a > b ? b : a; }

__device__ float Max(float a, float b) { return a > b ? a : b; }

__device__ Point projectionPointToLine(Point LP1, Point LP2, Point P) {
    Point result;
    if (LP2.x == LP1.x) {
        result.x = LP1.x;
        result.y = P.y;
    } else {
        // 如果该线段不平行于X轴也不平行于Y轴，则斜率存在且不为0。设线段的两端点为pt1和pt2，斜率为：
        float k = (LP2.y - LP1.y) / (LP2.x - LP1.x);
        // 该直线方程为:					y = k* ( x - pt1.x) + pt1.y
        // 其垂线的斜率为 -1/k,垂线方程为:	y = (-1/k) * (x - point.x) + point.y
        // 联立两直线方程解得:
        result.x = (k * k * LP1.x + k * (P.y - LP1.y) + P.x) / (k * k + 1);
        result.y = k * (result.x - LP1.x) + LP1.y;
    }
    return result;
}

__device__ bool if_finite(float a) {
    return fabs(a) < INITIAL_VALUE;
}

__device__ bool IsInField(Point p, float buffer=1000) {
    return (p.x > -PITCH_LENGTH / 2 + buffer && p.x < PITCH_LENGTH / 2 - buffer
            && p.y < PITCH_WIDTH / 2 - buffer && p.y > -PITCH_WIDTH / 2 + buffer);
}

__device__ bool IsInPenalty(Point p, float buffer=2000) {
    return (p.x < -PITCH_LENGTH/2 + PENALTY_LENGTH + buffer && p.x > -PITCH_LENGTH/2 && p.y > -PENALTY_WIDTH/2 - buffer && p.y < PENALTY_WIDTH/2 + buffer)
            || (p.x > PITCH_LENGTH/2 - PENALTY_LENGTH - buffer && p.x < PITCH_LENGTH/2 && p.y > -PENALTY_WIDTH/2 - buffer && p.y < PENALTY_WIDTH/2 + buffer);
}

__device__ void CUDA_compute_motion_1d(float x0, float v0, float v1, float a_max, float d_max, float v_max, float a_factor, float &traj_time)
{
    float traj_time_acc, traj_time_dec, traj_time_flat;
    traj_time_acc = traj_time_dec = traj_time_flat = 0.0;
    if ((x0 == 0 && v0 == v1) || !if_finite(x0) || !if_finite(v0) || !if_finite(v1)) {
        traj_time = 0;
        return;
    }

    a_max /= a_factor;
    d_max /= a_factor;

    float accel_time_to_v1 = fabs(v1 - v0) / a_max;
    float accel_dist_to_v1 = fabs((v1 + v0) / 2.0) * accel_time_to_v1;
    float decel_time_to_v1 = fabs(v0 - v1) / d_max;
    float decel_dist_to_v1 = fabs((v0 + v1) / 2.0) * decel_time_to_v1;

    float period = 1 / 40.0;

    if (v0 * x0 > 0 || (fabs(v0) > fabs(v1) && decel_dist_to_v1 > fabs(x0))) {
        float time_to_stop = fabs(v0) / (d_max);
        float x_to_stop = v0 * v0 / (2.0 * d_max);

        CUDA_compute_motion_1d(x0 + copysign(x_to_stop, v0), 0, v1, a_max * a_factor, d_max * a_factor, v_max, a_factor, traj_time);
        traj_time += time_to_stop;
        traj_time /= 1.25;
        return;
    }

    if (fabs(v0) > fabs(v1)) {
        traj_time_acc = (sqrt((d_max * v0 * v0 + a_max * (v1 * v1 + 2 * d_max * fabs(x0))) / (a_max + d_max)) - fabs(v0)) / a_max;

        if (traj_time_acc < 0.0)
            traj_time_acc = 0;
        traj_time_dec = ((fabs(v0) - fabs(v1)) + a_max * traj_time_acc) / d_max;
    }

    else if (accel_dist_to_v1 > fabs(x0)) {
        traj_time_acc = (sqrt(v0 * v0 + 2 * a_max * fabs(x0)) - fabs(v0)) / a_max;
        traj_time_dec = 0.0;
    }

    else {
        traj_time_acc = (sqrt((d_max * v0 * v0 + a_max * (v1 * v1 + 2 * d_max * fabs(x0))) / (a_max + d_max)) - fabs(v0)) / a_max;
        if (traj_time_acc < 0.0)
            traj_time_acc = 0;
        traj_time_dec = ((fabs(v0) - fabs(v1)) + a_max * traj_time_acc) / d_max;
    }


    if (traj_time_acc * a_max + fabs(v0) > v_max) {
        float dist_without_flat = (v_max * v_max - v0 * v0) / (2 * a_max) + (v_max * v_max - v1 * v1) / (2 * d_max);
        traj_time_flat = (fabs(x0) - dist_without_flat) / v_max;
    }
    else {
        traj_time_flat = 0;
    }

    if (FRAME_PERIOD * a_max + fabs(v0) > v_max && traj_time_flat > period) {
        traj_time = traj_time_flat + traj_time_dec;
    }
    else if (traj_time_acc < period && traj_time_dec == 0.0) {
        traj_time = traj_time_acc;
    }
    else if (traj_time_acc < period && traj_time_dec > 0.0) {
        traj_time = traj_time_dec;
    }
    else {
        traj_time = traj_time_acc + traj_time_flat / 1.1 + traj_time_dec / 1.1;
    }
}

__device__ float CUDA_predictedTime(float x0, float y0, float x1, float y1, float vx, float vy, bool isTheir) {
    float timeX = 0.0;
    float timeY = 0.0;
    float x = x0 - x1;
    float y = y0 - y1;

    float newVelAngle = atan2(vy, vx) - atan2(y, x);
    float velLength = sqrt(vx * vx + vy * vy);
    vx = velLength * cospi(newVelAngle / PI);
    vy = velLength * sinpi(newVelAngle / PI);

    x = sqrt(x * x + y * y);
    y = 0;

    float maxAcc, maxDec, maxSpeed;
    if(isTheir) {
        maxAcc = THEIR_MAX_ACC;
        maxDec = THEIR_MAX_DEC;
        maxSpeed = THEIR_MAX_SPEED;

    } else {
        maxAcc = OUR_MAX_ACC;
        maxDec = OUR_MAX_DEC;
        maxSpeed = OUR_MAX_SPEED;
    }
    CUDA_compute_motion_1d(x, vx, 0.0, maxAcc, maxDec, maxSpeed, 1.5, timeX);
    CUDA_compute_motion_1d(y, vy, 0.0, maxAcc, maxDec, maxSpeed, 1.5, timeY);
    if (timeX < 1e-5 || timeX > 50) timeX = 0;
    if (timeY < 1e-5 || timeY > 50) timeY = 0;
    return (timeX > timeY ? timeX : timeY);
}

__device__ bool CUDA_predictedInterTime(Point mePoint, Point ballPoint, Vector meVel, Vector ballVel, Point* interceptPoint, float* interTime, float responseTime, bool isTheir, float* rollingFraction, float* slidingFraction) {
    float ballSlidAcc = (*slidingFraction) / 2;
    float ballRollAcc = (*rollingFraction) / 2;
    static const float stepTime = 0.02;
    static const float PASS_VEL_DECAY = 5.0 / 7.0;
    static const float FIELD_BUFFER = 300.0;
    static const float PENALTY_BUFFER = 200.0;
    static const float AVOID_DIST = 4 * PLAYER_SIZE;
    // 初始化球速、加速度、球移动的距离、球初始位置
    const float originVel = sqrt(ballVel.x * ballVel.x + ballVel.y * ballVel.y);
    const float maxMoveTime = /*originVel * 2.0 / 7.0 / ballSlidAcc + */originVel * PASS_VEL_DECAY / ballRollAcc;
    float testVel = originVel;
    float ballAcc = ballSlidAcc;
    float testMoveDist = 0;
    float ballMoveTime = 0;
    Point testPoint = ballPoint;

///for debug
//    Vector velDir;
//    velDir.x = ballVel.x / originBallVel;
//    velDir.y = ballVel.y / originBallVel;
////    float meX = 547;
////    float meY = -276;
//    float meX = 564;
//    float meY = 159;

//    float vel = 236;
//    float dir = 4.41;
//    if(!isTheir && mePoint.x < meX + 5 && mePoint.x > meX - 5 && mePoint.y < meY + 5 && mePoint.y > meY - 5 && originBallVel > vel - 1.0 && originBallVel < vel + 1.0  && atan(ballVel.y / ballVel.x) > dir -  PI - 0.1 && atan(ballVel.y / ballVel.x) < dir -  PI + 0.1) {
//        printf("%lf %lf, %lf, %lf, interTime: %f\n", testPoint.x, testPoint.y, originBallVel, atan(ballVel.y / ballVel.x), ballSlidAcc);
//    }

    Vector ballDirec;
//    float slidingDist = 0, slidingTime = 0, slindingVel = 0;
    bool canInter = true, /*isSliding = true,*/ theirCanTouch = false;
    for (ballMoveTime = 0; ballMoveTime < maxMoveTime; ballMoveTime += stepTime ) {
//        if(isSliding) {
//            testVel = originVel - ballAcc * ballMoveTime;//v_0-at
//            if(testVel < 5.0 * originVel / 7.0) {
//                isSliding = false;
//                ballMoveTime = slidingTime = originVel * 2.0 / 7.0 / ballSlidAcc;
//                slidingDist = 12 * originVel * originVel / 49.0 / ballAcc;
//                testVel = slindingVel = originVel * 5.0 / 7.0;
//                ballAcc = ballRollAcc;
//            }
//            testMoveDist = PLAYER_CENTER_TO_BALL_CENTER + (originVel + testVel) * ballMoveTime / 2;
//        }
//        else {
//            testVel = slindingVel - ballAcc * (ballArriveTime - slidingTime);//v_0-at
//            testBallLength = PLAYER_CENTER_TO_BALL_CENTER + (slindingVel + testVel) * (ballArriveTime - slidingTime) / 2 + slidingDist;
//        }
        // 计算球速、球移动的距离
        testVel = originVel * PASS_VEL_DECAY - ballRollAcc * ballMoveTime;
        if(testVel < 0) testVel = 0;
        testMoveDist = (originVel * PASS_VEL_DECAY + testVel) * ballMoveTime / 2;
        // 计算截球点
        ballDirec.x = testMoveDist * ballVel.x / originVel;
        ballDirec.y = testMoveDist * ballVel.y / originVel;
        testPoint.x = ballPoint.x + ballDirec.x;
        testPoint.y = ballPoint.y + ballDirec.y;
        // 对敌方截球点进行特殊处理
        if(isTheir) {
            Vector adjustDir;
            adjustDir.x = mePoint.x - testPoint.x;
            adjustDir.y = mePoint.y - testPoint.y;
            float length = sqrt(adjustDir.x * adjustDir.x + adjustDir.y * adjustDir.y);
            adjustDir.x /= length;
            adjustDir.y /= length;
            testPoint.x += adjustDir.x * AVOID_DIST;
            testPoint.y += adjustDir.y * AVOID_DIST;
//            // 在球线上认为可以截球
//            if(sqrt((mePoint.x - testPoint.x) * (mePoint.x - testPoint.x) + (mePoint.y - testPoint.y) * (mePoint.y - testPoint.y)) < AVOID_DIST) {
//                theirCanTouch = true;
//                break;
//            }
        }
        // 计算截球时间
        float meArriveTime = CUDA_predictedTime(mePoint.x, mePoint.y, testPoint.x, testPoint.y, meVel.x, meVel.y, isTheir);

        if(IsInPenalty(testPoint, PENALTY_BUFFER))
            continue;
        if (!IsInField(testPoint, FIELD_BUFFER)) {
            canInter = false;
            break;
        }
        if(meArriveTime + responseTime < ballMoveTime){
            break;
        }
    }
//    if(testVel < 0)
//        printf("%lf, %lf, %lf, %lf \n", testVel, ballArriveTime, max_time, slidingTime);

//    if(originBallVel > 400 && ballArriveTime >= max_time)
//        printf("testBallLength: %lf  %lf   %lf\n", testBallLength, originBallVel, testBallLength);
    // 无法截球
    if(!canInter || (CAN_NOT_GET_STOP_BALL && ballMoveTime >= maxMoveTime)) {
        interceptPoint->x = INITIAL_VALUE;
        interceptPoint->y = INITIAL_VALUE;
        *interTime = INITIAL_VALUE;
        return false;
    }
    // 能够截球计算截球时间和截球点
    *interceptPoint = testPoint;
    *interTime = CUDA_predictedTime(mePoint.x, mePoint.y, interceptPoint->x, interceptPoint->y, meVel.x, meVel.y, isTheir);
//    if(theirCanTouch ) *interTime = 0.0;
    return true;
}

__device__ bool CUDA_predictedChipInterTime(Point mePoint, Point ballPoint, Vector meVel, Vector ballVel, Point* interceptPoint, float* interTime, float responseTime, bool isTheir, float* rollingFraction) {
    float chipVel = sqrt(ballVel.x * ballVel.x + ballVel.y * ballVel.y);
    float meArriveTime = INITIAL_VALUE;
    float ballAcc = (*rollingFraction) / 2.0;
    float ballAccSecondJump = 0;
    float stepTime = 0.05;
    float testBallLength = 0;
    Point testPoint = ballPoint;

    // 挑球第一段的时间, 单位s
    float time_1 = 2.0 * chipVel * sin(CHIP_FIRST_ANGLE) / 1000.0 / G;
    // 挑球第一段的距离, 单位m
    float length_1 = 1.0 / 2 * G * time_1 * time_1 / tan(CHIP_FIRST_ANGLE);
    // 挑球第二段的距离, 单位m
    float length_2 = (CHIP_LENGTH_RATIO - 1.0) * length_1;
    // 挑球第二段的时间, 单位s
    float time_2 = sqrt(2 * length_2 * tan(CHIP_SECOND_ANGLE) / G); // 单位s
    // 挑球第一二段的距离, 单位mm
    length_1 *= 1000;
    length_2 *= 1000;



    // 可以开始截球的起始距离
    float jumpDist = 0;
    // 可以开始截球的起始速度
    float moveVel = 0;
    // 可以开始截球的起始时刻
    float ballDropTime = 0;
    // 球滚动的最大时间
    float max_time = 0;
    if(isTheir) {
        jumpDist = length_1;
        moveVel = length_2 / time_2;
        ballDropTime = time_1;
        max_time = time_2 + chipVel * chipVel * CHIP_VEL_RATIO / 980 / ballAcc;

//        if(chipVel > 303 && chipVel < 305 && mePoint.x > 304 && mePoint.y < -240 && atan(ballVel.y / ballVel.x) > 5.74 - 2 * PI && atan(ballVel.y / ballVel.x) < 5.75 - 2 * PI) {
//            printf("%lf, %lf, %lf, %lf\n", moveVel, ballAccSecondJump, 2 * length_2 / time_2, time_2);
//        }
    } else {
        jumpDist = length_1 + length_2;
        moveVel = chipVel * chipVel * CHIP_VEL_RATIO / 980;
        ballDropTime = time_1 + time_2;
        max_time = moveVel / ballAcc;
    }

    bool canInter = true, theirCanTouch = false, isSecondJump = true;
    float afterArrivedTime = 0, secondJumpDist = 0, secondJumpTime = 0, secondJumpVelLeft = 0;
    while (afterArrivedTime < max_time) {
        Vector direc;
        if(isTheir) {
            if(isSecondJump) {
                testBallLength = jumpDist + moveVel * afterArrivedTime;
                if(testBallLength > length_1 + length_2) {
                    secondJumpDist = length_2;
                    secondJumpTime = time_2;
                    secondJumpVelLeft = chipVel * chipVel * CHIP_VEL_RATIO / 980;
                    isSecondJump = false;
                }
            }
            else {
                testBallLength = jumpDist + secondJumpDist + (secondJumpVelLeft * (afterArrivedTime - secondJumpTime) - 0.5 * ballAcc * (afterArrivedTime - secondJumpTime) * (afterArrivedTime - secondJumpTime));
            }
        }
        else {
            testBallLength = jumpDist + (moveVel * afterArrivedTime - 0.5 * ballAcc * afterArrivedTime * afterArrivedTime);
        }

        direc.x = testBallLength * ballVel.x / chipVel;
        direc.y = testBallLength * ballVel.y / chipVel;
        testPoint.x = ballPoint.x + direc.x;
        testPoint.y = ballPoint.y + direc.y;

        if(isTheir) {
            if(sqrt((mePoint.x - testPoint.x) * (mePoint.x - testPoint.x) + (mePoint.y - testPoint.y) * (mePoint.y - testPoint.y)) < PLAYER_SIZE * 1.2) {
                theirCanTouch = true;
                break;
            } else {
                Vector adjustDir;
                adjustDir.x = mePoint.x - testPoint.x;
                adjustDir.y = mePoint.y - testPoint.y;
                float length = sqrt(adjustDir.x * adjustDir.x + adjustDir.y * adjustDir.y);
                adjustDir.x /= length;
                adjustDir.y /= length;
                testPoint.x += adjustDir.x * PLAYER_SIZE;
                testPoint.y += adjustDir.y * PLAYER_SIZE;
            }
        }

        meArriveTime = CUDA_predictedTime(mePoint.x, mePoint.y, testPoint.x, testPoint.y, meVel.x, meVel.y, isTheir);

        if(meArriveTime < 0.10) meArriveTime = 0;

        if(IsInPenalty(testPoint, 200)) {
            afterArrivedTime += stepTime;
            continue;
        }
        if (!IsInField(testPoint)) {
            canInter = false;
            break;
        }
        if(meArriveTime + responseTime < ballDropTime + afterArrivedTime) {
            break;
        }
        afterArrivedTime += stepTime;
    }

    if(!canInter || (CAN_NOT_GET_STOP_BALL && afterArrivedTime >= max_time)){
        interceptPoint->x = INITIAL_VALUE;
        interceptPoint->y = INITIAL_VALUE;
        *interTime = INITIAL_VALUE;
        return false;
    }
    *interceptPoint = testPoint;
    *interTime = CUDA_predictedTime(mePoint.x, mePoint.y, testPoint.x, testPoint.y, meVel.x, meVel.y, isTheir);
    *interTime = max(*interTime, ballDropTime);

//    Vector velDir;
//    velDir.x = ballVel.x / chipVel;
//    velDir.y = ballVel.y / chipVel;
////    float meX = 547;
////    float meY = -276;
//    float meX = 444;
//    float meY = -105;

//    float vel = 428;
//    float dir = 4.07;
//    if(!isTheir && mePoint.x < meX + 5 && mePoint.x > meX - 5 && mePoint.y < meY + 5 && mePoint.y > meY - 5 && chipVel > vel - 1.0 && chipVel < vel + 1.0  && atan(ballVel.y / ballVel.x) > dir - 0.1 -  PI && atan(ballVel.y / ballVel.x) < dir + 0.1 - PI) {
//        printf("%lf %lf (%lf, %lf), (%lf, %lf), %lf, %lf, interTime: %f\n", testPoint.x, testPoint.y, ballPoint.x + velDir.x * length_1, ballPoint.y + velDir.y * length_1, ballPoint.x + velDir.x * (length_2 + length_1), ballPoint.y + velDir.y * (length_2 + length_1), chipVel, atan(ballVel.y / ballVel.x), *interTime);
//    }


    if(theirCanTouch){
        *interTime = 0.0;
    }

//    if((*interceptPoint).x > -85 && (*interceptPoint).x < -75 && (*interceptPoint).y < -5 && (*interceptPoint).y > -15)
//        printf("%lf \n", atan(ballVel.y / ballVel.x));

//    float vel = 202;
//    float dir = 3.28885;
//    if(isTheir && ballPoint.x + velDir.x * length_1 < 0 && chipVel > vel - 1.0 && chipVel < vel + 1.0  && atan(ballVel.y / ballVel.x) > dir - 0.001 - PI && atan(ballVel.y / ballVel.x) < dir + 0.001 - PI) {
//        printf("%lf, %lf, (%lf, %lf)\n", *interTime, testBallLength, (*interceptPoint).x, (*interceptPoint).y);
//    }

    return true;
}

// attack threat Evaluation Function for Run Pos
// # attack
// 1.距离对方球门的距离 2.射门有效角度 3.跟球的距离 4.对方车到传球线的距离 5.对方车到接球点的距离
// # defence
// !!!!!!!!!!!!!!!!!!!!!! 可以根據場上形式使用不同的公式
//__device__ float CUDA_evaluateFunc(Point candidate, Point ballPos, Player* enemy, Player receiver)
//{
//    float score = -INITIAL_VALUE;
//    // 1.距离对方球门的距离
//    Point goal;
//    goal.x = 600;
//    goal.y = 0;
//    float distToGoal = sqrt((candidate.x - goal.x) * (candidate.x - goal.x) + (candidate.y - goal.y) * (candidate.y - goal.y));

//    // 2.射门有效角度
//    Point leftGoalPost;
//    Point rightGoalPost;
//    leftGoalPost.x = rightGoalPost.x = 600;
//    leftGoalPost.y = -60;
//    rightGoalPost.y = 60;
//    float leftDir = atan2((candidate.y - leftGoalPost.y) , (candidate.x - leftGoalPost.x));
//    float rightDir = atan2((candidate.y - rightGoalPost.y) , (candidate.x - rightGoalPost.x));
//    float shootAngle = fabs(leftDir - rightDir);
//    shootAngle = shootAngle > PI ? 2*PI - shootAngle : shootAngle;

//    // 3.跟球的距离
//    float distToBall = sqrt((candidate.x - ballPos.x) * (candidate.x - ballPos.x) + (candidate.y - ballPos.y) * (candidate.y - ballPos.y));
//    // 4.对方车到传球线的距离
//    float distToPassLine = INITIAL_VALUE;
//    for (int i=0; i < MAX_PLAYER; i++) {
//        if(enemy[i].isValid){
//            Point projection = projectionPointToLine(candidate, ballPos, enemy[i].Pos);
//            // 判断是否在线段之间
//            if(projection.x > Min(ballPos.x, candidate.x) && projection.x < Max(ballPos.x, candidate.x)){
//                float dist = sqrt((projection.x - enemy[i].Pos.x) * (projection.x - enemy[i].Pos.x) + (projection.y - enemy[i].Pos.y) * (projection.y - enemy[i].Pos.y));
//                if(dist < distToPassLine)
//                    distToPassLine = dist;
//            }
//        }
//    }
//    // 5.对方车到接球点的距离
//    float distToEnemy = INITIAL_VALUE;
//    for (int i=0; i < MAX_PLAYER; i++) {
//        if(enemy[i].isValid){
//            float dist = sqrt((candidate.x - enemy[i].Pos.x) * (candidate.x - enemy[i].Pos.x) + (candidate.y - enemy[i].Pos.y) * (candidate.y - enemy[i].Pos.y));
//            if(dist < distToEnemy)
//                distToEnemy = dist;
//        }
//    }

//    // 当满足最低要求时计算得分
//    if(distToGoal < maxDistToGoal && shootAngle >= minShootAngle && distToBall < maxDistToBall
//            && distToPassLine >= minDistToPassLine && distToEnemy >= minDistToEnemy){
//        // 归一化处理
//        distToGoal = 1 - distToGoal/maxDistToGoal;
//        shootAngle = shootAngle/maxShootAngle;
//        distToBall = 1 - distToBall/maxDistToBall;
//        distToPassLine = distToPassLine/maxDistToPassLine;
//        distToEnemy = distToEnemy/maxDistToEnemy;

//        // 计算得分
//        score = weight1*distToGoal + weight2*shootAngle + weight3*distToBall + weight4*distToPassLine + weight5*distToEnemy;
//    }
//    return score;
//}

__global__ void calculateAllInterInfo(Player* players, Point* ballPos, rType* bestPass, float* rollingFraction, float* slidingFraction) {
    int angleIndex = threadIdx.x;
    int speedIndex = blockIdx.x;
    int playerNum =  blockIdx.y;

    int offset = blockIdx.y + gridDim.y * (threadIdx.x + blockIdx.x * blockDim.x);
    bool isTheir = playerNum < MAX_PLAYER ? false : true;
    float responseTime = isTheir ? THEIR_RESPONSE_TIME : OUR_RESPONSE_TIME;
    Vector ballVel;
    float interTime;
    Point interPoint;
//    if(playerNum == 2)
//        printf("pos: (%f, %f)\n", players[playerNum].Pos.x, players[playerNum].Pos.y);
//        printf("vel: (%f, %f)\n", players[playerNum].Vel.x, players[playerNum].Vel.y);
//    if(playerNum == 0)
//        printf("valid: %d\n", players[playerNum].isValid);

    ballVel.x = (speedIndex * BALL_SPEED_UNIT + MIN_BALL_SPEED) * cospi( 2.0 * angleIndex / THREAD_NUM_PASS);
    ballVel.y = (speedIndex * BALL_SPEED_UNIT + MIN_BALL_SPEED) * sinpi( 2.0 * angleIndex / THREAD_NUM_PASS);

    interTime = INITIAL_VALUE;
    interPoint.x = INITIAL_VALUE;
    interPoint.y = INITIAL_VALUE;

    if(players[playerNum].isValid)
         CUDA_predictedInterTime(players[playerNum].Pos, *ballPos, players[playerNum].Vel, ballVel, &interPoint, &interTime, responseTime, isTheir, rollingFraction, slidingFraction);
//    if(playerNum == 3 && interPoint.x > 512 && interPoint.y > -167 && interTime < 10)
//        printf("interTime: %f\n", interTime);

    bestPass[offset].interPos = interPoint;
    bestPass[offset].interTime = interTime;
    bestPass[offset].playerIndex = playerNum;
    bestPass[offset].dir = 2.0 * PI * angleIndex / THREAD_NUM_PASS;
    bestPass[offset].Vel = speedIndex * BALL_SPEED_UNIT + MIN_BALL_SPEED;


//    /***************** chip *******************/
    interTime = INITIAL_VALUE;
    interPoint.x = INITIAL_VALUE;
    interPoint.y = INITIAL_VALUE;
    ballVel.x = (speedIndex * CHIP_SPEED_UNIT + MIN_CHIP_SPEED) * cospi(2.0 * angleIndex / THREAD_NUM_PASS);
    ballVel.y = (speedIndex * CHIP_SPEED_UNIT + MIN_CHIP_SPEED) * sinpi(2.0 * angleIndex / THREAD_NUM_PASS);

    if(players[playerNum].isValid)
         CUDA_predictedChipInterTime(players[playerNum].Pos, *ballPos, players[playerNum].Vel, ballVel, &interPoint, &interTime, responseTime, isTheir, rollingFraction);

//        if(bestPass[offset].interPos.x > 299 && bestPass[offset].interPos.x < 300 && bestPass[offset].interPos.y > 248 && bestPass[offset].interPos.y < 249)
//            printf("%d\n", bestPass[offset].playerIndex);


    offset += BLOCK_X_PASS * BLOCK_Y_PASS * THREAD_NUM_PASS;
    bestPass[offset].interPos = interPoint;
    bestPass[offset].interTime = interTime;
    bestPass[offset].playerIndex = playerNum;
    bestPass[offset].dir = 2.0 * PI * angleIndex / THREAD_NUM_PASS;
    bestPass[offset].Vel = speedIndex * CHIP_SPEED_UNIT + MIN_CHIP_SPEED;
    __syncthreads();
}

__global__ void getBest(rType* passPoints) {
    __shared__ rType iP[BLOCK_Y_PASS];
    int blockId = blockIdx.y * gridDim.x + blockIdx.x;
    int playerNum = threadIdx.x;
    rType temp;
    bool even = true;
    int i;
    iP[playerNum] = passPoints[blockId * blockDim.x + playerNum];
    __syncthreads();
    for(i = 0; i < blockDim.x; i++) {
        if(playerNum < blockDim.x - 1 && even && iP[playerNum].interTime > iP[playerNum + 1].interTime) {
            temp = iP[playerNum + 1];
            iP[playerNum + 1] = iP[playerNum];
            iP[playerNum] = temp;
        }
        else if(playerNum > 0 && !even && iP[playerNum].interTime < iP[playerNum - 1].interTime) {
            temp = iP[playerNum];
            iP[playerNum] = iP[playerNum - 1];
            iP[playerNum - 1] = temp;
        }
        even = !even;
        __syncthreads();
    }
    passPoints[blockId * blockDim.x + playerNum] = iP[playerNum];
    __syncthreads();
    /************************/
    iP[playerNum] = passPoints[blockId * blockDim.x + playerNum + BLOCK_X_PASS * BLOCK_Y_PASS * THREAD_NUM_PASS];
    __syncthreads();
    even = true;
    for(i = 0; i < blockDim.x; i++) {
        if(playerNum < blockDim.x - 1 && even && iP[playerNum].interTime > iP[playerNum + 1].interTime) {
            temp = iP[playerNum + 1];
            iP[playerNum + 1] = iP[playerNum];
            iP[playerNum] = temp;
        }
        else if(playerNum > 0 && !even && iP[playerNum].interTime < iP[playerNum - 1].interTime) {
            temp = iP[playerNum];
            iP[playerNum] = iP[playerNum - 1];
            iP[playerNum - 1] = temp;
        }
        even = !even;
        __syncthreads();
    }

//    float interPointX = 370;
//    float interPointY = -108;

//    float meX = 444;
//    float meY = -105;

//    float vel = 428;
//    float dir = 4.07;
//    if(iP[playerNum].playerIndex == 1 && iP[playerNum].interPos.x < interPointX + 1.0 && iP[playerNum].interPos.x > interPointX - 1.0 && iP[playerNum].interPos.y < interPointY + 1.0 && iP[playerNum].interPos.y > interPointY - 1.0) {
//        printf("this: %f\n fast: %f\n fastIdx: %d \n\n", iP[playerNum].interTime, iP[playerNum - 2].interTime, iP[playerNum - 2].playerIndex);
//    }

    passPoints[blockId * blockDim.x + playerNum + BLOCK_X_PASS * BLOCK_Y_PASS * THREAD_NUM_PASS] = iP[playerNum];
    __syncthreads();

}

//__global__ void calculateAllPosScore(Player* Players, Point* ballPos, scoreAndPoint* allScore) {
//    float blockLength = PITCH_LENGTH / gridDim.x;
//    float blockWidth = PITCH_WIDTH / gridDim.y;
//    float threadLength = blockLength / blockDim.x;
//    float threadWidth = blockWidth / blockDim.y;
//    int blockIndex = gridDim.x * blockIdx.y + blockIdx.x;
//    int threadIndex = blockDim.x * threadIdx.y + threadIdx.x;
//    int allScoreIndex = blockIndex * blockDim.x * blockDim.y + threadIndex;
//    __syncthreads();
//    allScore[allScoreIndex].p.x = blockLength * blockIdx.x + threadLength / 2 + threadLength * threadIdx.x - PITCH_LENGTH / 2;
//    allScore[allScoreIndex].p.y = blockWidth * blockIdx.y + threadWidth / 2 + threadWidth * threadIdx.y - PITCH_WIDTH / 2;
//    __syncthreads();
//    if(IsInPenalty(allScore[allScoreIndex].p))
//        allScore[allScoreIndex].score = INITIAL_VALUE;
//    else
//        allScore[allScoreIndex].score = CUDA_evaluateFunc(allScore[allScoreIndex].p, *ballPos, Players, Players[MAX_PLAYER]);
//    __syncthreads();
//}

//__global__ void sortPosScore(scoreAndPoint *allScore) {
//    __shared__ scoreAndPoint scoreBlock[THREAD_X_FOR_POS_SCORE * THREAD_Y_FOR_POS_SCORE];
//    int blockIndex = gridDim.x * blockIdx.y + blockIdx.x;
//    int threadIndex = blockDim.x * threadIdx.y + threadIdx.x;
//    int allScoreIndex = blockIndex * blockDim.x * blockDim.y + threadIndex;
//    scoreAndPoint temp;
//    scoreBlock[threadIndex] = allScore[allScoreIndex];
//    __syncthreads();
//    //并行地按照从大到小的次序进行排列
//    bool even = true;
//    for(int i = 0; i < THREAD_X_FOR_POS_SCORE * THREAD_Y_FOR_POS_SCORE; i++) {
//        if(threadIndex < THREAD_X_FOR_POS_SCORE * THREAD_Y_FOR_POS_SCORE - 1 && even && scoreBlock[threadIndex].score < scoreBlock[threadIndex + 1].score) {
//            temp = scoreBlock[threadIndex + 1];
//            scoreBlock[threadIndex + 1] = scoreBlock[threadIndex];
//            scoreBlock[threadIndex] = temp;
//        }
//        else if(threadIndex > 0 && !even && scoreBlock[threadIndex].score > scoreBlock[threadIndex - 1].score) {
//            temp = scoreBlock[threadIndex];
//            scoreBlock[threadIndex] = scoreBlock[threadIndex - 1];
//            scoreBlock[threadIndex - 1] = temp;
//        }
//        even = !even;
//        __syncthreads();
//    }
//    allScore[allScoreIndex] = scoreBlock[threadIndex];
//    __syncthreads();
//}

extern "C" void BestPass(Player* players, Point* ball, rType* result, float* rollingFraction, float* slidingFraction) {
    rType *bestPass;

    hipMallocManaged((void**)&bestPass, 2 * BLOCK_X_PASS * BLOCK_Y_PASS * THREAD_NUM_PASS * sizeof(rType));

//    cudaEvent_t start, stop;
//    cudaEventCreate(&start);
//    cudaEventCreate(&stop);
//    cudaEventRecord(start);

    dim3 bolcks(BLOCK_X_PASS, BLOCK_Y_PASS);
    calculateAllInterInfo <<< bolcks, THREAD_NUM_PASS >>> (players, ball, bestPass, rollingFraction, slidingFraction);
    hipDeviceSynchronize();
    dim3 blocks2(BLOCK_X_PASS, THREAD_NUM_PASS);
    getBest<<< blocks2, BLOCK_Y_PASS >>> (bestPass);
    hipDeviceSynchronize();
    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess){
        printf("CUDA ERROR: %d\n", (int)cudaStatus);
        printf("Error Name: %s\n", hipGetErrorName(cudaStatus));
        printf("Description: %s\n", hipGetErrorString(cudaStatus));
    }

//    cudaEventRecord(stop);
//    cudaEventSynchronize(stop);

//    float milliseconds = 0;
//    cudaEventElapsedTime(&milliseconds, start, stop);
//    printf("Time: %.5f ms\n", milliseconds);

    rType defaultPlayer;
    defaultPlayer.dir = INITIAL_VALUE;
    defaultPlayer.interPos.x = INITIAL_VALUE;
    defaultPlayer.interPos.y = INITIAL_VALUE;
    defaultPlayer.interTime = INITIAL_VALUE;
    defaultPlayer.Vel = INITIAL_VALUE;
    defaultPlayer.deltaTime = -INITIAL_VALUE;
    defaultPlayer.playerIndex = INITIAL_VALUE;
    for(int i = 0; i < BLOCK_X_PASS * BLOCK_Y_PASS * THREAD_NUM_PASS; i += BLOCK_Y_PASS) {
        int playerNum = 0;
        for(int j = 0; j < MAX_PLAYER; j++) {
            if(bestPass[i + j].playerIndex >= MAX_PLAYER ) {
                while(playerNum < MAX_PLAYER) {
                    result[i / 2 + playerNum] = defaultPlayer;
                    playerNum++;
                }
                for(int k = 0; k < j; k++) {
                    result[i / 2 + k].deltaTime = bestPass[i + j].interTime - result[i / 2 + k].interTime;
                    if(result[i / 2 + k].deltaTime < MIN_DELTA_TIME)
                        result[i / 2 + k] = defaultPlayer;
                }
                break;
            }
            else {
                result[i / 2 + playerNum] = bestPass[i + j];
                playerNum++;
            }
        }
    }


    for(int i = BLOCK_X_PASS * BLOCK_Y_PASS * THREAD_NUM_PASS; i < 2 * BLOCK_X_PASS * BLOCK_Y_PASS * THREAD_NUM_PASS; i += BLOCK_Y_PASS) {
        int playerNum = 0;
        for(int j = 0; j < MAX_PLAYER; j++) {
//            float interPointX = 370;
//            float interPointY = -108;

//            float meX = 444;
//            float meY = -105;

//            float vel = 428;
//            float dir = 4.07;
//            if(bestPass[i + j].playerIndex == 1 && bestPass[i + j].interPos.x < interPointX + 1.0 && bestPass[i + j].interPos.x > interPointX - 1.0 && bestPass[i + j].interPos.y < interPointY + 1.0 && bestPass[i + j].interPos.y > interPointY - 1.0) {
//                printf("this: %f\n fast: %f\n fastIdx: %d \n\n", bestPass[i + j].interTime, bestPass[i + 1].interTime, bestPass[i + 1].playerIndex);
//            }

            if(bestPass[i + j].playerIndex >= MAX_PLAYER) {
                while(playerNum < MAX_PLAYER) {
                    result[i / 2 + playerNum] = defaultPlayer;
                    playerNum++;
                }
                for(int k = 0; k < j; k++) {
                    result[i / 2 + k].deltaTime = bestPass[i + j].interTime - result[i / 2 + k].interTime;

                    if(result[i / 2 + k].deltaTime < MIN_DELTA_TIME)
                        result[i / 2 + k] = defaultPlayer;
                }
                break;
            }
            else {
                result[i / 2 + playerNum] = bestPass[i + j];
                playerNum++;
            }
        }
    }
    hipFree(bestPass);
}

//extern "C" void PosScore(Player* players, Point* ballPos, Point* bestPositions) {
//    scoreAndPoint *allScore;
//    cudaMallocManaged((void**)&allScore, BLOCK_X_FOR_POS_SCORE * BLOCK_Y_FOR_POS_SCORE * THREAD_X_FOR_POS_SCORE * THREAD_Y_FOR_POS_SCORE * sizeof(scoreAndPoint));
////    cudaEvent_t start, stop;
////    cudaEventCreate(&start);
////    cudaEventCreate(&stop);
////    cudaEventRecord(start);

//    dim3 blocks(BLOCK_X_FOR_POS_SCORE, BLOCK_Y_FOR_POS_SCORE);
//    dim3 threads(THREAD_X_FOR_POS_SCORE, THREAD_Y_FOR_POS_SCORE);
//    calculateAllPosScore<<< blocks, threads >>> (players, ballPos, allScore);
//    cudaDeviceSynchronize();

//    sortPosScore<<< blocks, threads >>> (allScore);
//    cudaDeviceSynchronize();

//    cudaError_t cudaStatus = cudaGetLastError();
//    if (cudaStatus != cudaSuccess){
//        printf("CUDA ERROR: %d\n", (int)cudaStatus);
//        printf("Error Name: %s\n", cudaGetErrorName(cudaStatus));
//        printf("Description: %s\n", cudaGetErrorString(cudaStatus));
//    }
////    cudaEventRecord(stop);
////    cudaEventSynchronize(stop);
////    float milliseconds = 0;
////    cudaEventElapsedTime(&milliseconds, start, stop);
////    printf("Time: %.5f ms\n", milliseconds);

//    for(int i = 0; i < BLOCK_X_FOR_POS_SCORE * BLOCK_Y_FOR_POS_SCORE; i++) {
//        bestPositions[i] = allScore[i * THREAD_X_FOR_POS_SCORE * THREAD_Y_FOR_POS_SCORE].p;
////        printf("(%lf, %lf)\n", allScore[i * THREAD_X_FOR_POS_SCORE * THREAD_Y_FOR_POS_SCORE].p.x, allScore[i * THREAD_X_FOR_POS_SCORE * THREAD_Y_FOR_POS_SCORE].p.y);
//    }
//    cudaFree(allScore);
//    return;
//}
